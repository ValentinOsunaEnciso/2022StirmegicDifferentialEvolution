#include "hip/hip_runtime.h"
﻿/*Differential Evolution, version DE/best/1/bin; Valentin Osuna-Enciso, DIC/2020, UDG*/
#include "interface.h"

//FUNCIONES NUEVAS; SE NOMBRAN COMO APARECEN EN [2015Dogon]:
//Stepint; D>2; limits={-5.12,5.12}; f*=25-6*D; x*=(-5.12<=x_i<-5); Unimodal; Separable.
__device__ double F1(double* x) {
	double suma = 0.0;
	for (int i = 0; i < D; i++) {
		suma += ceil(x[i]);
	}
	return 25 + suma;
}
__device__ double* F1lims() {
	double limites[] = { -5.12, 5.12 };
	return limites;
}
//Step; D>2; limits={-100,100}; f*=0; Unimodal; Separable.
__device__ double F2(double* x) {
	double suma = 0.0;
	for (int i = 0; i < D; i++) {
		suma += pow(floor(x[i] + 0.5), 2.0);
	}
	return suma;
}
__device__ double* F2lims() {
	double limites[] = { -100.0, 100.0 };
	return limites;
}
//Sphere; D>2; limits={-100,100}; f*=0; Unimodal; Separable.
__device__ double F3(double* x) {
	double sum = 0.0;
	for (int i = 0; i < D; i++)
		sum += pow(x[i], 2.0);
	return sum;
}
__device__ double* F3lims() {
	double limites[] = { -5.12, 5.12 };
	return limites;
}
//Sumsquares; D>2; limits={-10,10}; f*=0; Unimodal; Separable.
__device__ double F4(double* x) {
	double sum = 0.0;
	for (int i = 0; i < D; i++)
		sum += double(i) * pow(x[i], 2.0);
	return sum;
}
__device__ double* F4lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}
//Trid; D>2; limits={-D^2,D^2}; f*=-D(D+4)(D-1)/6; Unimodal; No separable.
__device__ double F5(double* x) {
	double sum1 = pow(x[0] - 1, 2.0), sum2 = 0.0;
	for (int i = 1; i < D; i++) {
		sum1 += pow(x[i] - 1, 2.0);
		sum2 += x[i] * x[i - 1];
	}
	return sum1 - sum2;
}
__device__ double* F5lims() {
	double limites[] = { -pow(double(D),2.0), pow(double(D),2.0) };
	return limites;
}
//Zakharov; D>2; limites={-5,10}; f*=0; Unimodal; No separable.
__device__ double F6(double* x) {
	double sum1 = 0.0, sum2 = 0.0;
	for (int i = 0; i < D; i++) {
		sum1 += pow(x[i], 2.0);
		sum2 += 0.5 * (double(i) + 1.0) * x[i];
	}
	return sum1 + pow(sum2, 2.0) + pow(sum2, 4.0);
}
__device__ double* F6lims() {
	double limites[] = { -5.0, 10.0 };
	return limites;
}
//Schwefel 2.22; D>2; limits={-10,10}; f*=0; Unimodal; No separable.
__device__ double F7(double* x) {
	double suma = 0.0, multiplicacion = 1.0;
	for (int i = 0; i < D; i++) {
		suma += abs(x[i]);
		multiplicacion *= abs(x[i]);
	}
	return suma + multiplicacion;
}
__device__ double* F7lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}
//Schwefel 1.2; D>2; limits={-10,10}; f*=0; Unimodal; No separable.
__device__ double F8(double* x) {
	double suma = 0.0, suma2 = 0.0;
	for (int i = 0; i < D; i++) {
		suma2 = 0.0;
		for (int j = 0; j < i; j++) {
			suma2 += x[j];
		}
		suma += pow(suma2, 2.0);
	}
	return suma;
}
__device__ double* F8lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}
//Rosenbrock; D>2; limites={-30,30}; f*=0; Unimodal; No separable.
__device__ double F9(double* x) {
	double sum = 0.0;
	for (int i = 0; i < D - 1; i++)
		sum += 100.0 * pow(x[i + 1] - pow(x[i], 2.0), 2.0) + pow(x[i] - 1.0, 2.0);
	return sum;
}
__device__ double* F9lims() {
	double limites[] = { -5.0, 10.0 };
	return limites;
}
//Dixon-Price; D>2; limites={-10,10}; f*=0; Unimodal; No separable.
__device__ double F10(double* x) {
	double sum = 0.0;
	for (int i = 1; i < D; i++)
		sum += (double(i) + 1.0) * pow(2 * pow(x[i], 2.0) - x[i - 1], 2.0);
	return sum + pow(x[0] - 1.0, 2.0);
}
__device__ double* F10lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}

//Rastrigin; D>2; limites={-5.12,5.12}; f*=0; Multimodal; Separable.
__device__ double F11(double* x) {
	double sum = 0.0;
	for (int i = 0; i < D; i++)
		sum += pow(x[i], 2.0) - 10.0 * cos(2.0 * PI * x[i]);
	return sum + 10.0 * D;
}
__device__ double* F11lims() {
	double limites[] = { -5.12, 5.12 };
	return limites;
}
//Schwefel; D>2; limits={-500,500}; f*=-418.9829*D; Multimodal; Separable.
__device__ double F12(double* x) {
	double f = 0.0;
	for (int i = 0; i < D; i++)
		f -= x[i] * sin(sqrt(fabs(x[i])));
	f += 418.9829 * D;
	return f;
}
__device__ double* F12lims() {
	double limites[] = { -500.0, 500.0 };
	return limites;
}
//Griewank; D>2; limits={-600,600}; f*=0; Multimodal; No separable.
__device__ double F13(double* x) {
	double sum = 0.0, prod = 1.0;
	for (int i = 0; i < D; i++) {
		sum += pow(x[i], 2.0) / 4000.0;
		prod = prod * cos(x[i] / sqrt(i + 1.0));
	}
	return sum - prod + 1.0;
}
__device__ double* F13lims() {
	double limites[] = { -600.0, 600.0 };
	return limites;
}
//Ackley; D>2; limits={-32.768, 32.768}; f*=0; Multimodal; No separable.
__device__ double F14(double* x) {
	double sum1 = 0.0, sum2 = 0.0, a = 20.0, b = 0.2, c = 2 * PI;
	for (int i = 0; i < D; i++) {
		sum1 += pow(x[i], 2.0);
		sum2 += cos(c * x[i]);
	}
	return -a * exp(-b * sqrt(sum1 / D)) - exp(sum2 / D) + a + exp(1.0);
}
__device__ double* F14lims() {
	double limites[] = { -32.768, 32.768 };
	return limites;
}
//Penalized1; D>2; limits={-50,50}; f*=0; Multimodal; No separable.
__device__ double F15(double* x) {
	double sum1 = 0.0, sum2 = 0.0;
	for (int i = 0; i < D - 1; i++) {
		sum1 += pow(x[i] - 1.0, 2.0) * (1.0 + 10.0 * pow(sin(PI * x[i + 1]), 2.0));
	}
	sum1 += 10.0 * pow(sin(PI * x[0]), 2.0) + pow(x[-1] - 1.0, 2.0);
	sum1 = (sum1 * PI) / D;
	for (int i = 0; i < D; i++) {
		if (x[i] > 10.0)
			sum2 += 100.0 * pow(x[i] - 10.0, 4.0);
		else if (x[i] < -10.0)
			sum2 += 100.0 * pow(-x[i] - 10.0, 4.0);
	}
	return sum1 + sum2;
}
__device__ double* F15lims() {
	double limites[] = { -50.0, 50.0 };
	return limites;
}
//Penalized2; D>2; limits={-50,50}; f*=0; Multimodal; No separable.
__device__ double F16(double* x) {
	double sum1 = 0.0, sum2 = 0.0;
	for (int i = 0; i < D - 1; i++) {
		sum1 += pow(x[i] - 1.0, 2.0) * (1.0 + 10.0 * pow(sin(3.0 * PI * x[i + 1]), 2.0));
	}
	sum1 += pow(sin(PI * x[0]), 2.0) + pow(x[-1] - 1.0, 2.0) * (1.0 + pow(sin(2.0 * PI * x[-1]), 2.0));

	for (int i = 0; i < D; i++) {
		if (x[i] > 5.0)
			sum2 += 100.0 * pow(x[i] - 10.0, 4.0);
		else if (x[i] < -5.0)
			sum2 += 100.0 * pow(-x[i] - 10.0, 4.0);
	}
	return sum1 + sum2;
}
__device__ double* F16lims() {
	double limites[] = { -50.0, 50.0 };
	return limites;
}
//Levy; D>2; limits={-10,10}; f*=0.
__device__ double F17(double* x) {
	double sum = 0.0, term1, term3, prod = 1.0, w[D];
	int i;
	for (i = 0; i < D; i++)
		w[i] = 1.0 + (x[i] - 1.0) / 4.0;
	term1 = pow(sin(PI * w[0]), 2.0);
	term3 = pow(w[D - 1] - 1.0, 2.0) * (1.0 + pow(sin(2.0 * PI * w[D - 1]), 2.0));
	for (i = 0; i < D - 1; i++)
		sum += pow(w[i] - 1, 2.0) * (1.0 + 10.0 * pow(sin(PI * w[i] + 1.0), 2.0));
	return term1 + sum + term3;
}
__device__ double* F17lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}
//Perm; D>2; limits={-D,D}; f*=0.
__device__ double F18(double* x) {
	double outer = 0.0, inner;
	for (int i = 0; i < D; i++) {
		inner = 0.0;
		for (int j = 0; j < D; j++)
			inner += (double(j) + 1.0 + 10.0) * (pow(x[j], double(i) + 1.0) - pow(1.0 / (double(j) + 1.0), double(i) + 1.0));
		outer += pow(inner, 2.0);
	}
	return outer;
}
__device__ double* F18lims() {
	double limites[] = { -double(D), double(D) };
	return limites;
}
//Hyperellipsoid; D>2; limits={-65.53,65.53}; f*=0.
__device__ double F19(double* x) {
	double outer = 0.0, inner;
	for (int i = 0; i < D; i++) {
		inner = 0.0;
		for (int j = 0; j < D; j++)
			inner += pow(x[j], 2.0);
		outer += inner;
	}
	return outer;
}
__device__ double* F19lims() {
	double limites[] = { -65.536, 65.536 };
	return limites;
}
//Sum Power; D>2; limits={-1,1}, f*=0.
__device__ double F20(double* x) {
	double sum = 0.0;
	for (int i = 0; i < D; i++)
		sum += pow(fabs(x[i]), double(i) + 2.0);
	return sum;
}
__device__ double* F20lims() {
	double limites[] = { -1.0, 1.0 };
	return limites;
}

//Michalewicz2; D=2; limits={0,PI}; f*=-1.8013; Multimodal; Separable.
__device__ double F21(double* x) {
	double suma = 0.0, m = 10.0;
	suma = sin(x[0]) * pow(sin(pow(x[0], 2.0) / PI), 2.0 * m) + sin(x[1]) * pow(sin(pow(x[1], 2.0) / PI), 2.0 * m);
	return suma;
}
__device__ double* F21lims() {
	double limites[] = { 0.0, PI };
	return limites;
}
//Schaffer; D=2; limits={-100,100}; f*=0; Multimodal; No separable.
__device__ double F22(double* x) {
	double fact1 = 0.0, fact2 = 0.0;
	fact1 = pow(sin(pow(x[0], 2.0) - pow(x[1], 2.0)), 2.0) - 0.5;
	fact2 = pow(1.0 + 0.001 * (pow(x[0], 2.0) + pow(x[1], 2.0)), 2.0);
	return 0.5 + fact1 / fact2;
}
__device__ double* F22lims() {
	double limites[] = { -100.0, 100.0 };
	return limites;
}
//Six Hump Camel Back; D=2; limits={-5,5}; f*=-1.0316; Multimodal; No separable.
__device__ double F23(double* x) {
	double suma = 0.0;
	suma = 4.0 * pow(x[0], 2.0) - 2.1 * pow(x[0], 4.0) + (1.0 / 3.0) * pow(x[0], 6.0) + x[0] * x[1] - 4.0 * pow(x[1], 2.0) + 4.0 * pow(x[1], 4.0);
	return suma;
}
__device__ double* F23lims() {
	double limites[] = { -5.0, 5.0 };
	return limites;
}
//Bohachevsky2; D=2; limits={-100,100}; f*=0; Multimodal; No separable.
__device__ double F24(double* x) {
	double suma = 0.0;
	suma = pow(x[0], 2.0) + 2.0 * pow(x[1], 2.0) - 0.3 * cos(3.0 * PI * x[0]) * (4.0 * PI * x[1]) + 0.3;
	return suma;
}
__device__ double* F24lims() {
	double limites[] = { -100.0, 100.0 };
	return limites;
}
//Bohachevsky3; D=2; limits={-100,100}; f*=0; Multimodal; No separable.
__device__ double F25(double* x) {
	double suma = 0.0;
	suma = pow(x[0], 2.0) + 2.0 * pow(x[1], 2.0) - 0.3 * cos(3.0 * PI * x[0] + 4.0 * PI * x[1]) + 0.3;
	return suma;
}
__device__ double* F25lims() {
	double limites[] = { -100.0, 100.0 };
	return limites;
}
//Shubert; D=2; limits={-10,10}; f*=-186.7309; Multimodal; No separable.
__device__ double F26(double* x) {
	double fact1 = 0.0, fact2 = 0.0;
	for (int i = 0; i < 5; i++) {
		fact1 += (i + 1.0) * cos((i + 2.0) * x[0] + (i + 1.0));
		fact2 += (i + 1.0) * cos((i + 2.0) * x[1] + (i + 1.0));
	}
	return fact1 * fact2;
}
__device__ double* F26lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}
//Goldstein-Price; D=2; limits={-2,2}; f*=3.0; Multimodal; No separable.
__device__ double F27(double* x) {
	double fact1a = 0.0, fact1b = 0.0, fact1c = 0.0, fact2a = 0.0, fact2b = 0.0, fact2c = 0.0;
	fact1a = pow(x[0] + x[1] + 1.0, 2.0);
	fact1b = 19.0 - 14.0 * x[0] + 3.0 * pow(x[0], 2.0) - 14.0 * x[1] + 6.0 * x[0] * x[1] + 3.0 * pow(x[1], 2.0);
	fact1c = 1.0 + fact1a * fact1b;
	fact2a = pow(2.0 * x[0] - 3.0 * x[1], 2.0);
	fact2b = 18.0 - 32.0 * x[0] + 12.0 * pow(x[0], 2.0) + 48.0 * x[1] - 36.0 * x[0] * x[1] + 27.0 * pow(x[1], 2.0);
	fact2c = 30.0 + fact2a * fact2b;
	return fact1c * fact2c;
}
__device__ double* F27lims() {
	double limites[] = { -2.0, 2.0 };
	return limites;
}
//Langermann; D=2; limits={0,10}; f*=-1.4; Multimodal; No separable.
__device__ double F28(double* x) {
	double suma1 = 0.0, fact1 = 0.0, fact2 = 0.0, suma2 = 0.0;
	double a1[] = { 3.0, 5.0, 2.0, 1.0, 7.0 };
	double a2[] = { 5.0, 2.0, 1.0, 4.0, 9.0 };
	double c[] = { 1.0, 2.0, 5.0, 2.0, 3.0 };
	for (int i = 0; i < 5; i++) {
		suma1 = pow(x[0] - a1[i], 2.0) + pow(x[1] - a2[i], 2.0);
		fact1 = c[i] * exp(-(1.0 / PI) * suma1);
		fact2 = cos(PI * suma1);
		suma2 += fact1 * fact2;
	}
	return suma2;
}
__device__ double* F28lims() {
	double limites[] = { 0.0, 10.0 };
	return limites;
}

//Beale; D=2; limits={-4.5,4.5}; f*=0; Unimodal; No separable.
__device__ double F29(double* x) {
	double suma = pow((1.5 - x[0] * (1.0 - x[1])), 2.0) + pow((2.25 - x[0] * (1.0 - pow(x[1], 2.0))), 2.0) + pow((2.625 - x[0] * (1.0 - pow(x[1], 3.0))), 2.0);
	return suma;
}
__device__ double* F29lims() {
	double limites[] = { -4.5, 4.5 };
	return limites;
}
//Easom; D=2; limits={-100,100}; f*=-1.0; Unimodal; No separable. 
__device__ double F30(double* x) {
	double suma = -cos(x[0]) * cos(x[1]) * exp(-pow(x[0] - PI, 2.0) - pow(x[1] - PI, 2.0));
	return suma;
}
__device__ double* F30lims() {
	double limites[] = { -100.0, 100.0 };
	return limites;
}
//Matyas; D=2; limits={-10,10}; f*=0; Unimodal; No separable.
__device__ double F31(double* x) {
	double suma = 0.26 * (pow(x[0], 2.0) + pow(x[1], 2.0)) - 0.48 * x[0] * x[1];
	return suma;
}
__device__ double* F31lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}
//Foxholes; D=2; limits = {-65.536, 65.536}; f*=0.9980038; Multimodal; Separable.
__device__ double F32(double* x) {
	double suma = 0.0, suma2 = 0.0;
	double a1[] = { -32.0, -16.0, 0.0, 16.0, 32.0, -32.0, -16.0, 0.0, 16.0, 32.0, -32.0, -16.0, 0.0, 16.0, 32.0, -32.0, -16.0, 0.0, 16.0, 32.0, -32.0, -16.0, 0.0, 16.0, 32.0 };
	double a2[] = { -32.0, -32.0, -32.0, -32.0, -32.0, -16.0, -16.0, -16.0, -16.0, -16.0, 0.0, 0.0, 0.0, 0.0, 0.0, 16.0, 16.0, 16.0, 16.0, 16.0, 32.0, 32.0, 32.0, 32.0, 32.0 };
	for (int j = 0; j < 25; j++) {
		suma2 = 1.0 / (j + 1 + pow(x[0] - a1[j], 6.0) + pow(x[1] - a2[j], 6.0));
		suma += suma2;
	}
	suma2 = (1.0 / 500.0) + suma;
	suma = pow(suma2, -1.0);
	return suma;
}
__device__ double* F32lims() {
	double limites[] = { -65.536, 65.536 };
	return limites;
}
//Bohachevsky1; D=2; limits={-100,100}; f*=0; Multimodal; Separable.
__device__ double F33(double* x) {
	double suma = 0.0;
	suma = pow(x[0], 2.0) + 2.0 * pow(x[1], 2.0) - 0.3 * cos(3.0 * PI * x[0]) - 0.4 * cos(4.0 * PI * x[1]) + 0.7;
	return suma;
}
__device__ double* F33lims() {
	double limites[] = { -100.0, 100.0 };
	return limites;
}
//Booth; D=2; limits={-10,10}; f*=0; Multimodal; Separable.
__device__ double F34(double* x) {
	double suma = 0.0;
	suma = pow(x[0] + 2.0 * x[1] - 7.0, 2.0) + pow(2.0 * x[0] + x[1] - 5, 2.0);
	return suma;
}
__device__ double* F34lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}
//Holder; D=2; limits={-10,10}; f*=-19.2085; x*=(8.05502,9.66459); Multimodal; No separable.
__device__ double F35(double* x) {
	double suma = 0.0;
	suma = abs(sin(x[0]) * cos(x[1]) * exp(abs(1.0 - (sqrt(pow(x[0], 2.0) + pow(x[1], 2.0)) / PI))));
	return suma;
}
__device__ double* F35lims() {
	double limites[] = { -10.0, 10.0 };
	return limites;
}
__device__ int* shuffleR1R2(double* x, int indiceHilo) {
	int r1 = Np * x[indiceHilo * 2], r2 = Np * x[indiceHilo * 2 + 1], cont = 2;
	while ((r1 == r2) && cont < 2 * Np) {
		r1 = __double2int_ru(Np * x[indiceHilo + cont + 1]);
		r2 = __double2int_ru(Np * x[indiceHilo + cont++]);
	}
	int limites[] = { r1, r2 };
	return limites;
}

//Lineas clave para llamadas dinamicas a funciones:
typedef double (*llamadaaFunciones)(double*);
__device__ llamadaaFunciones funciones[] = { F1, F2, F3, F4, F5, F6, F7, F8, F9, F10, F11, F12, F13, F14, F15, F16, F17, F18, F19, F20, F21, F22, F23, F24, F25, F26, F27, F28, F29, F30, F31, F32, F33, F34, F35 };
typedef double* (*llamadaaFunciones2)();
__device__ llamadaaFunciones2 limites[] = { F1lims, F2lims, F3lims, F4lims, F5lims, F6lims, F7lims, F8lims, F9lims, F10lims, F11lims, F12lims, F13lims, F14lims, F15lims, F16lims, F17lims, F18lims, F19lims, F20lims, F21lims, F22lims, F23lims, F24lims, F25lims, F26lims, F27lims, F28lims, F29lims, F30lims, F31lims, F32lims, F33lims, F34lims, F35lims };

__global__ void evaluaFuncion(double* x, double* out, int funcion) {
	//evalua funciones; version global memory.
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int id = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	out[id] = funciones[funcion](&x[id * D]);
	__syncthreads();
}

__global__ void f1b(double* x, double* out) {
	//Schwefel; f*=-41898.29; lims=[-500,500]; x*=[420.9687,...]
	//version shared memory.
	int blockId = blockIdx.y * gridDim.x + blockIdx.x;
	int i, id = blockId * blockDim.x + threadIdx.x;
	__shared__ double compartida[D];
	for (i = 0; i < D; i++) compartida[id + i] = x[id * D + i];
	//__syncthreads();
	double resultado = 0.0;
	for (i = id; i < id + D; i++)
		resultado -= compartida[i] * sin(sqrt(fabs(compartida[i])));
	out[id] = resultado;
}

__global__ void iniciaX(double* X, double* A, double* inferior, double* superior) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int id = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	X[id] = inferior[id] + (superior[id] - inferior[id]) * A[id];
}

__global__ void iniciaL(double* inferior, double* superior, int funcion) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int id = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	double* x;
	x = limites[funcion]();
	inferior[id] = x[0];
	superior[id] = x[1];
}

//############### Busca el minimo global:  #################################################
//##(Taken from: http://supercomputingblog.com/cuda/cuda-tutorial-3-thread-communication/)##
__global__ void getMinGlobal(double* F, double* fbest, double* xbest, double* X, int* indices) {
	__shared__ double min[Np];		// Declare array to be in shared memory.
	__shared__ int indexes[Np];		//pInd is array of indexs where a min is found
	// Calculate which element this thread reads from memory	
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int arrayIndex = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x, i;
	min[threadIdx.x] = F[arrayIndex];
	indexes[threadIdx.x] = indices[arrayIndex];
	__syncthreads();
	int nTotalThreads = blockDim.x;	// Total number of active threads
	//printf("HILO: %d\n", threadIdx.x);
	while (nTotalThreads > 1) {
		int halfPoint = (nTotalThreads >> 1);	// divide by two
		// only the first half of the threads will be active.
		if (threadIdx.x < halfPoint) {
			// Get the shared value stored by another thread
			double temp = min[threadIdx.x + halfPoint];
			//printf("temp: %f\n", temp);
			int temp1 = indexes[threadIdx.x + halfPoint];
			//printf("hilo: %d; medio: %d, %f y %f; %d\n", threadIdx.x, halfPoint, min[threadIdx.x], temp, blockDim.x);
			if (temp < min[threadIdx.x]) {
				min[threadIdx.x] = temp;
				indexes[threadIdx.x] = temp1;
			}
		}
		__syncthreads();
		nTotalThreads = (nTotalThreads >> 1);	// divide by two.
	}
	// At this point in time, thread zero has the min
	// It's time for thread zero to write it's final results.
	// Note that the address structure of pResults is different, because
	// there is only one value for every thread block.
	if (threadIdx.x == 0) {
		fbest[blockIdx.y + blockIdx.x] = min[0];
		for (i = 0; i < D; i++) { xbest[i] = X[(indexes[0]) * D + i]; }
	}
	__syncthreads();
}

__global__ void generaEvaluaU(double* aleat1, double* aleat2, double* X, double* U, double* FU, double* xbest, double f, double cr, int funcion, double* li, double* ls) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int arrayIndex = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	//int r1 = Np * aleat1[arrayIndex * 3], r2 = Np * aleat1[arrayIndex * 3 + 1], jrand = D * aleat1[arrayIndex * 3 + 2];
	int r1, r2, i, jrand = __double2int_ru(D * aleat1[arrayIndex * 3 + 2]);
	int* x;
	x = shuffleR1R2(aleat1, arrayIndex);
	r1 = x[0];
	r2 = x[1];
	//printf("HILO: %d; r1: %d, r2: %d; jrand: %d\n", arrayIndex, r1, r2, jrand);
	for (i = 0; i < D; i++) {
		if (aleat2[arrayIndex * D + i] < cr || i == jrand) {
			U[arrayIndex * D + i] = xbest[i] + f * (X[r1 * D + i] - X[r2 * D + i]);
			if (U[arrayIndex * D + i] > ls[arrayIndex * D + i])
				U[arrayIndex * D + i] = ls[arrayIndex * D + i] * aleat2[arrayIndex * D + i];
			if (U[arrayIndex * D + i] < li[arrayIndex * D + i])
				U[arrayIndex * D + i] = li[arrayIndex * D + i] * aleat2[arrayIndex * D + i];
		}
		else {
			U[arrayIndex * D + i] = X[arrayIndex * D + i];
		}
		//printf("%f\t", xbest[i]);
	}
	FU[arrayIndex] = funciones[funcion](&U[arrayIndex * D]);
	__syncthreads();
}

__global__ void comparaFUyF(double* X, double* U, double* F, double* FU) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int arrayIndex = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x, i;
	if (FU[arrayIndex] < F[arrayIndex]) {
		for (i = 0; i < D; i++)
			X[arrayIndex * D + i] = U[arrayIndex * D + i];
		F[arrayIndex] = FU[arrayIndex];
	}
	__syncthreads();
}

extern "C" void generaEvaluaMutantes(double* d_aleat3, double* d_aleat, double* d_X, double* d_U, double* d_FU, double* d_xbest, float F, float Cr, int funcion, double* d_liminf, double* d_limsup) {
	dim3 blocksXgridX(Np, 1);  		//Bloques en grid;Hilos por bloque
	generaEvaluaU << < 1, blocksXgridX >> > (d_aleat3, d_aleat, d_X, d_U, d_FU, d_xbest, F, Cr, funcion, d_liminf, d_limsup);
}

extern "C" void obtieneMinimoGlobal(double* d_F, double* d_fbest, double* d_xbest, double* d_X, int* d_ind) {
	dim3 blocksXgridX(Np, 1);  		//Bloques en grid;Hilos por bloque
	getMinGlobal << <1, blocksXgridX >> > (d_F, d_fbest, d_xbest, d_X, d_ind);
}

extern "C" void evaluaFitnessPoblacion(double* d_X, double* d_F, int funcion) {
	dim3 blocksXgridX(Np, 1);  		//Bloques en grid;Hilos por bloque
	evaluaFuncion << < blocksXgridX, 1 >> > (d_X, d_F, funcion);
}

extern "C" void iniciaPoblacionyLimites(double* d_X, double* d_aleat, double* d_liminf, double* d_limsup, int funcion) {
	dim3 blocksXgridX(Np, 1), hilosXblockX(D, 1);  		//Bloques en grid;Hilos por bloque
	iniciaL << <blocksXgridX, hilosXblockX >> > (d_liminf, d_limsup, funcion);
	iniciaX << <blocksXgridX, hilosXblockX >> > (d_X, d_aleat, d_liminf, d_limsup);
}

extern "C" void comparaFitnessUyX(double* d_X, double* d_U, double* d_F, double* d_FU) {
	dim3 blocksXgridX(Np, 1), hilosXblockX(D, 1);  		//Bloques en grid;Hilos por bloque
	comparaFUyF << < 1, blocksXgridX >> > (d_X, d_U, d_F, d_FU);
}
